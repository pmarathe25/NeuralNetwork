#include "hip/hip_runtime.h"
#include "Matrix.hpp"
#include "Layer/FullyConnectedLayer.hpp"

namespace ai{
    template <typename Matrix, float (*activationFunc)(float), float (*activationDeriv)(float)>
    FullyConnectedLayer<Matrix, activationFunc, activationDeriv>::FullyConnectedLayer(int inputSize, int outputSize) {
        this -> weights = Matrix(inputSize, outputSize);
        this -> biases = Matrix(1, outputSize);
        initializeWeights();
    }

    template <typename Matrix, float (*activationFunc)(float), float (*activationDeriv)(float)>
    FullyConnectedLayer<Matrix, activationFunc, activationDeriv>::FullyConnectedLayer(Matrix weights, Matrix biases) {
        this -> weights = weights;
        this -> biases = biases;
    }

    template <typename Matrix, float (*activationFunc)(float), float (*activationDeriv)(float)>
    Matrix FullyConnectedLayer<Matrix, activationFunc, activationDeriv>::feedForward(const Matrix& input) {
        return activate(getWeightedOutput(input));
    }

    template <typename Matrix, float (*activationFunc)(float), float (*activationDeriv)(float)>
    Matrix FullyConnectedLayer<Matrix, activationFunc, activationDeriv>::getWeightedOutput(const Matrix& input) {
        return (input * weights).addVector(biases);
    }

    template <typename Matrix, float (*activationFunc)(float), float (*activationDeriv)(float)>
    Matrix FullyConnectedLayer<Matrix, activationFunc, activationDeriv>::activate(const Matrix& weightedOutput) {
        return weightedOutput.template applyFunction<activationFunc>();
    }

    // Backpropagation for other layers.
    template <typename Matrix, float (*activationFunc)(float), float (*activationDeriv)(float)>
    Matrix FullyConnectedLayer<Matrix, activationFunc, activationDeriv>::backpropagate(const Matrix& input, const Matrix& intermediateDeltas, const Matrix& weightedOutput, float learningRate) {
        // Compute this layer's deltas
        Matrix deltas = intermediateDeltas.hadamard(weightedOutput.template applyFunction<activationDeriv>());
        // Use these deltas and then compute an intermediate quantity for the previous layer.
        return backpropagate(input, deltas, learningRate);
    }

    template <typename Matrix, float (*activationFunc)(float), float (*activationDeriv)(float)>
    void FullyConnectedLayer<Matrix, activationFunc, activationDeriv>::initializeWeights() {
        double weightRange = 2 / sqrt(weights.numRows());
        if (activationFunc == static_cast<float (*)(float)>(relu<float>)) {
            weights = Matrix::randomUniformLike(weights, 0, weightRange);
            biases = Matrix::randomNormalLike(biases, 0, weightRange).template applyFunction<abs>();
        } else {
            weights = Matrix::randomUniformLike(weights, -weightRange, weightRange);
            biases = Matrix::randomNormalLike(biases, 0, weightRange);
        }
    }

    // Processes deltas and computes a quantity for the previous layer.
    template <typename Matrix, float (*activationFunc)(float), float (*activationDeriv)(float)>
    Matrix FullyConnectedLayer<Matrix, activationFunc, activationDeriv>::backpropagate(const Matrix& input, const Matrix& deltas, float learningRate) {
        // For the previous layer.
        Matrix intermediateDeltas = deltas * weights.transpose();
        // Modify this layer's weights.
        weights -= input.transpose() * deltas * learningRate;
        // Return an intermediate quantity for the previous layer.
        return intermediateDeltas;
    }

    template class FullyConnectedLayer<Matrix_F, ai::sigmoid, ai::sigmoid_prime>;
    template class FullyConnectedLayer<Matrix_F, ai::relu, ai::relu_prime>;

} /* namespace ai */
